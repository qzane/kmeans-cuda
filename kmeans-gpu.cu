#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#define MAXN 1000000

//todo: read clusters from file
//todo: the choise for init clusters
//todo: the ending criteria

const int ThreadsPerBlock = 1024; // max value since CC2.0
int BlocksPerGrid = 0;

int N; // number of points
int K; // number of clusters
int T; // number of iterations
char INPUT_FILE[256]; // input file name

float *POINTS; // POINTS[i*2+0]:x POINTS[i*2+1]:y
int *CLASSES; // class for each point
int *NUM_CLASSES; // number of points in each class
float *CLUSTERS; // position for each cluster

// values on CUDA device

float *D_POINTS; // POINTS[i*2+0]:x POINTS[i*2+1]:y
int *D_CLASSES; // class for each point
int *D_NUM_CLASSES; // number of points in each class
float *D_CLUSTERS; // position for each cluster


void write_results(int n, int k){
    FILE *outputFile;
    int i;
    
    outputFile = fopen("Classes.txt", "w");
    for(i=0;i<n;i++){
        fprintf(outputFile, "%d\n", CLASSES[i]);
    }
    fclose(outputFile);
    
    outputFile = fopen("Clusters.txt", "w");
    for(i=0;i<k;i++){
        fprintf(outputFile, "%f,%f\n", CLUSTERS[i*2], CLUSTERS[i*2+1]);
    }
    fclose(outputFile);    
}

void update_classes(int n, int k){ //based on CLUSTERS
    int i,j,minK;
    float minDis, dis, disX, disY;
    for(i=0;i<n;i++){
        disX = POINTS[i*2]-CLUSTERS[0];
        disY = POINTS[i*2+1]-CLUSTERS[1];
        minK = 0;
        minDis = disX*disX + disY*disY;
        for(j=1;j<k;j++){
            disX = POINTS[i*2]-CLUSTERS[j*2];
            disY = POINTS[i*2+1]-CLUSTERS[j*2+1];
            dis = disX*disX + disY*disY;
            if(dis<minDis){
                minK = j;
                minDis = dis;
            }
        }
        CLASSES[i] = minK;
    }
}
    
void update_clusters(int n, int k){ // based on CLASSES
    int i,j,class;
    for(i=0;i<k;i++){
        CLUSTERS[i*2]=0;
        CLUSTERS[i*2+1]=0;
        NUM_CLASSES[i]=0;
    }
    for(i=0;i<n;i++){
        class = CLASSES[i];
        NUM_CLASSES[class]++;
        CLUSTERS[class*2] += POINTS[i*2];
        CLUSTERS[class*2+1] += POINTS[i*2+1];
    }
    for(i=0;i<k;i++){
        //if(NUM_CLASSES[i]!=0){
            CLUSTERS[i*2] /= NUM_CLASSES[i]; // produce nan when divided by 0
            CLUSTERS[i*2+1] /= NUM_CLASSES[i];
        //}
    }    
}
    
void clean_clusters(int *K){ // remove empty clusters, CLASSES are invalid after this process
    int i;
    float tmp;
    for(i=0;i<*K;i++){
        if(NUM_CLASSES[i]==0){
            CLUSTERS[i*2] = CLUSTERS[*K * 2];
            CLUSTERS[i*2+1] = CLUSTERS[*K * 2 + 1];
            *K--;
            i--; // the new cluster is not tested
        }
    }
}

void init(int n, int k, char *input){ // malloc and read points (and clusters)
    FILE *inputFile;
    int i;
    float x,y;
    
    // read points
    POINTS = (float*)malloc(n * 2 * sizeof(float));
    inputFile = fopen(input, "r");
    for(i=0;i<n;i++){
        if(fscanf(inputFile, "%f,%f\n", &x, &y)==2){
            POINTS[i*2] = x;
            POINTS[i*2+1] = y;
        }
    }
    fclose(inputFile);
    
    // classes init
    CLASSES = (int*)malloc(n * sizeof(int));
    
    // clusters init
    NUM_CLASSES = (int*)malloc(k * sizeof(int));
    CLUSTERS = (float*)malloc(k * 2 * sizeof(float));
    for(i=0;i<k;i++){
        CLUSTERS[i*2]=POINTS[i*2];
        CLUSTERS[i*2+1]=POINTS[i*2+1];
    }    
}


void cuda_init(int n, int k){ // malloc and copy data to device
	hipError_t err = hipSuccess;
	
	// malloc
	err &= hipMalloc((void **)&D_POINTS, sizeof(POINTS));
	err &= hipMalloc((void **)&D_CLASSES, sizeof(CLASSES));
	err &= hipMalloc((void **)&D_NUM_CLASSES, sizeof(NUM_CLASSES));
	err &= hipMalloc((void **)&D_CLUSTERS, sizeof(CLUSTERS));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	// copy data
	err = hipSuccess;
	err &= hipMemcpy(D_POINTS, POINTS, sizeof(POINTS), hipMemcpyHostToDevice);
	err &= hipMemcpy(D_CLUSTERS, CLUSTERS, sizeof(CLUSTERS), hipMemcpyHostToDevice);
	
	// blocksPerGrid
	BlocksPerGrid = (n + ThreadsPerBlock - 1) / ThreadsPerBlock;
	printf("Using %d blocks of %d threads\n", BlocksPerGrid, ThreadsPerBlock);
	
}


int data_count(char *fileName){
    FILE *inputFile;
    float x, y;
    int count=0;
    inputFile = fopen(fileName, "r");
    while(fscanf(inputFile, "%f,%f\n", &x, &y)==2){
        count++;    
        //printf("%f,%f\n",tmp1,tmp2);
    }
    fclose(inputFile);
    return count;
}


int cmd_parser(int argc, char **argv, int *n, int *k, int *t, char *input){
    int invalid;
    int valid;
    char ch;
    char usage[] = "Usage: %s -n N -k K -t T -i Input.txt\n"
                   "    N: Number_of_Points, default: the number of lines in Input_File\n"
                   "    K: default: 2\n"
                   "    T: max iterations for the kmeans algorithm\n"
                   "    Input: should be n lines, two floats in each line and split by ','\n"
		           "    Results will be in Classes.txt and Clusters.txt\n";
    invalid = 0;
    valid = 0;
    if(argc==1){
        invalid = 1;
    }
    
    //default values
    *n = -1;
    *k = 2;
    *t = 1;
    
    while((ch = getopt(argc, argv, "n:k:t:i:h")) != -1) {
        switch(ch) {
            case 'n':
                sscanf(optarg, "%d", n);
                break;
            case 'k':
                sscanf(optarg, "%d", k);
                break;
            case 't':
                sscanf(optarg, "%d", t);
                break;
            case 'i':
                strncpy(input, optarg, 256);
                valid = 1;
                break;
            case 'h':  //print help
                invalid = 1;
                break;
            case '?':
                invalid = 1;
            default:
                ;
        }
    }
	
    
    if(valid && *n==-1){
        *n = data_count(input);
    }
    
    
    invalid = invalid || !valid;
    if(invalid){
        printf(usage, argv[0]);
    }
    
    if(*n>MAXN){
        invalid = 1;
        printf("N is too large\n");
    }
    
    
    //printf("option N: %d\n", *n);
    //printf("option K: %d\n", *k);
    //printf("option T: %d\n", *t);
    //printf("option Input: %s\n", input);
    //printf("invalid %d\n", invalid);
	
    return invalid;    
}


int main(int argc, char **argv) {
    int t;
    if(cmd_parser(argc, argv, &N, &K, &T, INPUT_FILE)){ // not enough parameters
        return 1;
    }
    init(N, K, INPUT_FILE);
	update_classes(N, K);
    for(t=0;t<T;t++){
		if(t!=0){
			clean_clusters(&K);
		}
        update_classes(N, K);
        update_clusters(N, K);
    }
	
    write_results(N, K);
    return 0;
}
